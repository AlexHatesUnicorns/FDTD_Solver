#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include "Parser.h"
#include "../Updaters/UpdaterTE.h"
#include "../Grids/GridTE.h"
#include "../Routines/HzOutputRoutineTE.h"

#include <ctime>
#include <cmath>

int main(int argc, char **argv)
{
	/*int sizeX = 500;
	int sizeY = 500;
	int lambda = 100;
	GridTE grid(sizeX, sizeY, 1/sqrt(2));
	//grid.readEpsilon("single_particle"+std::to_string(sizeX-2000)+".txt");
	UpdaterTE updater;
	HarmonicSource hSource2(1, 4.44063/(float)(lambda), sizeX/4, sizeY/2);
	HzOutputRoutineTM Hzout("Hzout", &grid, 0, 500, 10, 0, 500, 10, 0, 1000, 20);
	updater.addGrid(&grid);
	updater.addSource(&hSource2);
	updater.addRoutine(&Hzout);
	unsigned int start_time =  clock(); // начальное время
	for(int i = 0; i < 2000; i++) {
		updater.iterate();
		if(i%10==0){
			std::cout<<"step :" << i << std::endl;
		}
	}
    unsigned int end_time = clock(); // конечное время
    unsigned int time = end_time - start_time; // искомое время
    std::cout <<"step per second : "<< 1000/((float)(time)/CLOCKS_PER_SEC) << std::endl;*/
	Parser parser;
	parser.runFile(argv[1]);

	return 0;
}
