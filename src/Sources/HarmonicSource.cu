#include "hip/hip_runtime.h"
/*
 * HarmonicSource.cpp
 *
 *  Created on: 11 янв. 2016 г.
 *      Author: aleksandr
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <thrust/device_ptr.h>
#include "thrust/for_each.h"
#include <thrust/iterator/counting_iterator.h>

#include "HarmonicSource.h"

void HarmonicSource::updateField(d_ptr _field, const int time) {
	updater.field = _field;
	thrust::counting_iterator<int> start(time);
	thrust::counting_iterator<int> end(time+1);
	thrust::for_each(start , end , updater);
}

__device__
void HarmonicUpdater::operator()(const int time) {
	field[0] = amp*sinf(freq*(float)(time));
}
