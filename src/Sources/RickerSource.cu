#include "hip/hip_runtime.h"
#include "RickerSource.h"

#include <iostream>
#include <thrust/device_ptr.h>
#include "thrust/for_each.h"
#include <thrust/iterator/counting_iterator.h>
#include "hip/hip_math_constants.h"
#include "math.h"

#include "RickerSource.h"

#define Pi HIP_PI_F

void RickerSource::updateField(d_ptr _field, const int time) {
	updater.field = _field;
	thrust::counting_iterator<int> start(time);
	thrust::counting_iterator<int> end(time+1);
	thrust::for_each(start , end , updater);
}

__device__
void RickerUpdater::operator()(const int time) {
	float temp = powf(Pi*(freq*time-M),2);
	field[0] = amp*(1-2*temp)*expf(-1.0f*temp);
}
